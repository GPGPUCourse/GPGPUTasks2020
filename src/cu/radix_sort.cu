#include "hip/hip_runtime.h"
#include <libgpu/cuda/cu/opencl_translator.cu>

#include "../cl/radix_sort.cl"

void cuda_get_inverse_bit(const gpu::WorkSize &workSize,
                          unsigned int* as, unsigned int* bit_array, const unsigned int n, const unsigned int bit,
                          hipStream_t stream)
{
    get_inverse_bit<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(as, bit_array, n, bit);
    CUDA_CHECK_KERNEL(stream);
}

void cuda_prefix_sum(const gpu::WorkSize &workSize,
                     unsigned int* partial_sum, unsigned int* prefix_sum_ptr, const unsigned int n, const unsigned int pow,
                     hipStream_t stream)
{
    prefix_sum<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(partial_sum, prefix_sum_ptr, n, pow);
    CUDA_CHECK_KERNEL(stream);
}

void cuda_partial_sum(const gpu::WorkSize &workSize,
                      unsigned int* cur_partial_sum, unsigned int* next_partial_sum, const unsigned int n,
                      hipStream_t stream)
{
    partial_sum<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(cur_partial_sum, next_partial_sum, n);
    CUDA_CHECK_KERNEL(stream);
}

void cuda_radix(const gpu::WorkSize &workSize,
                unsigned int* cur_as, unsigned int* next_as, unsigned int* prefix_sum, const unsigned int n, const unsigned int bit,
                hipStream_t stream)
{
    radix<<<workSize.cuGridSize(), workSize.cuBlockSize(), 0, stream>>>(cur_as, next_as, prefix_sum, n, bit);
    CUDA_CHECK_KERNEL(stream);
}
